
#include "hip/hip_runtime.h"


#include <stdio.h>
#include <stdlib.h>
#include <string.h>
#include <time.h>


void ising(int* G, double *w, int k, int n) {

	int* oldG = (int*)malloc(n * n * sizeof(int));
	for (int i = 0; i < n * n; i++) {
		oldG[i] = G[i];
	}
	int* newG = (int*)malloc(n * n * sizeof(int));
	double newSpin = 0;
	int* temp;
	//For every timestep
	for (int timestep = 0; timestep < k; timestep++) {

		//for every element
		for (int i = 0; i < n; i++) {
			for (int j = 0; j < n; j++) {

				//get spin of element from the others around it
				for (int ii = -2; ii <= 2; ii++) {
					for (int jj = -2; jj <= 2; jj++) {

						newSpin += w[(jj + 2) + (ii + 2) * 5] * oldG[((jj + j + n) % n) + ((i + ii + n) % n) * n];
					}
				}
				//In newSpin now there is the sum of all spins*weights of all nearby elements

				//if newSpin > 0 then the updated spin = 1
				if (newSpin > 0.000001) {
					newG[j + i * n] = 1;
				}
				//if newSpin < 0 then the updated spin = -1
				else if (newSpin < -0.000001) {
					newG[j + i * n] = -1;
				}
				//if newSpin = 0 then the updated spin = old spin
				else {
					newG[j + i * n] = oldG[j + i * n];
				}
				//reset newSpin
				newSpin = 0;
			}
		}
		//swap newG with G
		temp = newG;
		newG = oldG;
		oldG = temp;
		//Now new data are stored in oldG and we can overwrite newG

	}

	for (int i = 0; i < n * n; i++) {
		G[i] = oldG[i];
	}

}



int main()
{	
	clock_t start, end;
	double cpu_time;
	double w[25] = { 0.004,0.016,0.026 ,0.016 ,0.004 ,0.016,0.071,0.117,0.071,0.016,0.026,0.117,0,0.117, 0.026,0.016, 0.071 , 0.117 ,0.071 ,0.016 ,0.004 ,0.016 ,0.026 ,0.016 ,0.004 };
	int sum = 0;
	int n = 517;
	int k =1;
	int* G = (int*)malloc(n * n * sizeof(int));
	int* cG = (int*)malloc(n * n * sizeof(int));
	
	for (int i = 0; i < n * n; i++) {
		if (rand() % 2 == 0) {
			G[i] = 1;
		}
		else
			G[i] = -1;
	}

	
	memcpy(cG, G, n * n * sizeof(int));

	
	ising(G, w, k, n);
	
	

	//start = clock();
	//cising(cG, w, k, n);
	//end = clock();
	//cpu_time = ((double)(end - start)) / CLOCKS_PER_SEC;

	for (int i = 0; i < n * n; i++) {
		if (G[i] != cG[i])
			sum++;
	}
	printf("\nErrors: %d\n", sum);
	
	
	//printf("Time with %d moments and %d timesteps: %f seconds\n", n, k, cpu_time);
    return 0;
}