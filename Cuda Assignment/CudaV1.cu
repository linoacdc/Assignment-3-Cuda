#include "hip/hip_runtime.h"

#include "hip/hip_runtime.h"
#include ""

#include <stdio.h>





__global__ void cudaising(int* G, double* w, int* newG) {

	int index = threadIdx.x + blockIdx.x * blockDim.x;
	double newSpin = 0.0;
	for (int ii = -2; ii <= 2; ii++) {
		for (int jj = -2; jj <= 2; jj++) {

			newSpin += w[(jj + 2) + (ii + 2) * 5] * G[((jj + threadIdx.x + blockDim.x) % blockDim.x) + ((blockIdx.x + ii + blockDim.x) % blockDim.x) * blockDim.x];
		}
	}


	if (newSpin > 0.000001) {
		newG[index] = 1;
	}
	//if newSpin < 0 then the updated spin = -1
	else if (newSpin < -0.000001) {
		newG[index] = -1;
	}
	//if newSpin = 0 then the updated spin = old spin
	else {
		newG[index] = G[index];
	}


}

void cising(int* G, double* w, int k, int n) {

	int* d_G, * d_nG, * temp;
	double* d_w;
	int size = n * n * sizeof(int);
	hipMalloc((void**)&d_G, size);
	hipMalloc((void**)&d_w, 5 * 5 * sizeof(double));
	hipMalloc((void**)&d_nG, size);

	hipMemcpy(d_G, G, size, hipMemcpyHostToDevice);
	hipMemcpy(d_w, w, 5 * 5 * sizeof(double), hipMemcpyHostToDevice);
	//For every timestep
	for (int timestep = 0; timestep < k; timestep++) {

		cudaising <<< n, n >> > (d_G, d_w, d_nG);

		//swap newG with G
		temp = d_nG;
		d_nG = d_G;
		d_G = temp;
		//Now new data are stored in oldG and we can overwrite newG
	}
	hipMemcpy(G, d_G, size, hipMemcpyDeviceToHost);
	hipFree(d_G); hipFree(d_nG); hipFree(d_w);
}
