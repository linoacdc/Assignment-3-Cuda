#include "hip/hip_runtime.h"

#include "hip/hip_runtime.h"
#include ""

#include <stdio.h>



__global__ void cudaising(int* G, double* w, int* newG, int workperthread, int n) {

	//global id of first element whose spin is to be computed by this thread
	int startingId = (threadIdx.x + blockIdx.x * blockDim.x) * workperthread;
	int element = 0;
	//for each element which will be calculated by this thread
	for (int element = 0; element < workperthread; element++) {
		//calculate spin of element with global id = starting id + element
		double newSpin = 0.0;
		for (int ii = -2; ii <= 2; ii++) {
			for (int jj = -2; jj <= 2; jj++) {

				newSpin += w[(jj + 2) + (ii + 2) * 5] * G[((jj + threadIdx.x * workperthread + n + element) % n) + ((blockIdx.x + ii + n) % n) * n];
			}
		}
		int index = startingId + element;
		if (newSpin > 0.000001) {
			newG[index] = 1;
		}
		//if newSpin < 0 then the updated spin = -1
		else if (newSpin < -0.000001) {
			newG[index] = -1;
		}
		//if newSpin = 0 then the updated spin = old spin
		else {
			newG[index] = G[index];
		}

	}
}



void cising(int* G, double* w, int k, int n) {
	//
	int* d_G, * d_nG, * temp;
	double* d_w;
	int size = n * n * sizeof(int);
	hipMalloc((void**)&d_G, size);
	hipMalloc((void**)&d_w, 5 * 5 * sizeof(double));
	hipMalloc((void**)&d_nG, size);

	hipMemcpy(d_G, G, size, hipMemcpyHostToDevice);
	hipMemcpy(d_w, w, 5 * 5 * sizeof(double), hipMemcpyHostToDevice);
	//For every timestep
	for (int timestep = 0; timestep < k; timestep++) {
		//cudaising with 517 blocks and 47 threads each and 11 moments calculated per thread
		cudaising << < n, 47 >> > (d_G, d_w, d_nG, 11, n);

		//swap newG with G
		temp = d_nG;
		d_nG = d_G;
		d_G = temp;
		//Now new data are stored in oldG and we can overwrite newG
	}
	//copy updated matrix to host
	hipMemcpy(G, d_G, size, hipMemcpyDeviceToHost);
	hipFree(d_G); hipFree(d_nG); hipFree(d_w);


}
