#include "hip/hip_runtime.h"

#include "hip/hip_runtime.h"
#include ""
#include <stdio.h>




__global__ void cudaising(int* G, double* w, int* newG, int n, int workperthread) {

	int startingId = threadIdx.x * workperthread;

	//shared w and G in block
	__shared__ double tempW[5 * 5];
	__shared__ int tempG[(517 + 4) * 5];

	//copy necessary elements from G into tempG
	for (int i = -2; i <= 2; i++) {
		for (int j = -2; j <= n + 2; j++) {
			tempG[(j + 2) + (i + 2) * (n + 4)] = G[((j + n) % n) + ((blockIdx.x + i + n) % n) * n];
		}
	}


	//copy using threads
	/*if (threadIdx.x >=25&&threadIdx.x <30) {
		for (int j = -2; j <= n + 2; j++) {
			tempG[(j + 2) + (threadIdx.x-2-25 + 2) * (n + 4)] = G[((j + n) % n) + ((blockIdx.x + threadIdx.x-2-25 + n) % n) * n];

		}
	}
	*/


	//Copy w in tempW


	if (threadIdx.x < 25) {
		tempW[threadIdx.x] = w[threadIdx.x];
	}
	__syncthreads();




	//for every element computed by this thread
	for (int element = 0; element < workperthread; element++) {

		double newSpin = 0.0;

		//for every point in matrix w
		for (int ii = 0; ii < 5; ii++) {
			for (int jj = 0; jj < 5; jj++) {

				//compute new Spin of element
				newSpin += tempW[(jj)+(ii) * 5] * tempG[startingId + element + jj + ii * (n + 4)];

			}
		}
		//global index of element whose spin was just calculated
		int index = startingId + element + blockIdx.x * blockDim.x * workperthread;
		//if newSpin > 0 then the updated spin = 1
		if (newSpin > 0.000001) {
			newG[index] = 1;
		}

		//if newSpin < 0 then the updated spin = -1
		else if (newSpin < -0.000001) {
			newG[index] = -1;
		}

		//if newSpin = 0 then the updated spin = old spin
		else {
			newG[index] = G[index];
		}

	}
	__syncthreads();
}



void cising(int* G, double* w, int k, int n) {

	//device variables
	int* d_G, * d_nG, * temp;
	double* d_w;
	int size = n * n * sizeof(int);
	//Allocating space in device for the matrices
	hipMalloc((void**)&d_G, size);
	hipMalloc((void**)&d_w, 5 * 5 * sizeof(double));
	hipMalloc((void**)&d_nG, size);
	//copying the matrices in device
	hipMemcpy(d_G, G, size, hipMemcpyHostToDevice);
	hipMemcpy(d_w, w, 5 * 5 * sizeof(double), hipMemcpyHostToDevice);
	//For every timestep
	for (int timestep = 0; timestep < k; timestep++) {
		//run cudaising with 517 blocks, NOT threads each, NOE elements for each thread
		cudaising << < n, 47 >> > (d_G, d_w, d_nG, n, 11);
		//swap newG with G
		temp = d_nG;
		d_nG = d_G;
		d_G = temp;
		//Now new data are stored in oldG and we can overwrite newG

	}
	//copying final matrix to host
	hipMemcpy(G, d_G, size, hipMemcpyDeviceToHost);
	hipFree(d_G); hipFree(d_nG); hipFree(d_w);


}
